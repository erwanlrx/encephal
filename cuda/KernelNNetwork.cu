#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <vector>

#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "CudaType.h"

#define SHAREDSIZE ((16 * 1024)/sizeof(DTYPE))

extern __device__ void __syncthreads();

__global__ void kernelBuffer_propagation(buffer * buffers) {
	int id_buf = blockIdx.x;
	int id_coord = threadIdx.x;

	buffers[id_buf].propagation_valid[id_coord] = buffers[id_buf].propagation_calculated[id_coord];
}

__device__ void propagation_perceptron(buffer * input, buffer * output, PCPTR * perceptron) {
	int id = threadIdx.x;
	if (id < input->size)
		output->propagation_calculated[id] = ((DTYPE)1) / (1 + exp(-input->propagation_valid[id] - perceptron->bias[id]));
}
__device__ void propagation_fullconnection(buffer * input, buffer * output, FC * fconnection) {
	int id = threadIdx.x;
	__shared__ DTYPE input_buffer[SHAREDSIZE];
	int size_output = output->size;
	int size_input = input->size;
	if (id < size_input) {
		input_buffer[id] = input->propagation_valid[id];
		__syncthreads();
		DTYPE resultat = input_buffer[0] * fconnection->matrix[id * size_output];
		for (int i = 1; i < size_input; i++)
			resultat += input_buffer[i] * fconnection->matrix[id * size_output + i];

		output->propagation_calculated[id] = resultat;
	}
}
__global__ void kernelNetwork_propagation(buffer * buffers, node * nodes, FC * fc, PCPTR * pcptr) {
	int layer = blockIdx.x;
	buffer * input = buffers + layer;
	buffer * output = input + 1;

	switch (nodes[layer].type) {
	case PERCEPTRON:
		propagation_perceptron(input, output, pcptr + nodes[layer].indice);
		break;
	case FCONNECTION:
		propagation_fullconnection(input, output, fc + nodes[layer].indice);
		break;
	}
}

void propagation_kernelcall(int node_number, int maxdim, buffer * GPUbuffer_tab, node *GPUnode_tab, FC *GPUFC_tab, PCPTR *GPUPCPTR_tab) {
	kernelNetwork_propagation<<<node_number, maxdim>>>(GPUbuffer_tab, GPUnode_tab, GPUFC_tab, GPUPCPTR_tab);
	kernelBuffer_propagation<<<node_number + 1, maxdim>>>(GPUbuffer_tab);
}

__device__ void fullconnection_propagationNoPipeline(buffer * input, buffer * output, FC * fc) {
	int id = threadIdx.x;
	uint32_t input_size = input->size;
	uint32_t output_size = output->size;
	
	__shared__ DTYPE input_buffer[SHAREDSIZE];
	if (id < input_size)
		input_buffer[id % input_size] = input->propagation_valid[id % input_size];

	__syncthreads();

	if (id < output_size) {
		
		DTYPE resultat = input_buffer[0] * fc->matrix[id];
		for (uint32_t i = 1; i < input_size; i++) {
			resultat += input_buffer[i] * fc->matrix[id + output_size * i];
		}

		output->propagation_valid[id] = resultat;
	}
}
__device__ void perceptron_propagationNoPipeline(buffer * input, buffer * output, PCPTR * pcptr) {
	int id = threadIdx.x ;
	if (id < input->size)
		output->propagation_valid[id] = ((DTYPE)1) / (((DTYPE)1) + ((DTYPE)__expf(-input->propagation_valid[id] - pcptr->bias[id])));
}
__device__ void propagationNoPipeline(buffer * input, buffer * output, node * node, FC * fc, PCPTR * pcptr) {
	switch (node->type) {
	case PERCEPTRON:
		perceptron_propagationNoPipeline(input, output, &(pcptr[node->indice]));
		break;
	case FCONNECTION:
		fullconnection_propagationNoPipeline(input, output, &(fc[node->indice]));
		break;
	}
} 

__device__ void backpropagation_perceptron(buffer * input, buffer * output, PCPTR * node) {
	int id = threadIdx.x;
	if (id < input->size) {
		output->back_propagation[id] = input->back_propagation[id] * input->propagation_valid[id] * ((DTYPE)1 - input->propagation_valid[id]);
	}
}
__device__ void backpropagation_fullconnection(buffer * input, buffer * output, FC * node) {
	int id = threadIdx.x;
	__shared__ DTYPE input_buffer[SHAREDSIZE];
	int size_output = output->size;
	int size_input = input->size;

	if (id < size_input)
		input_buffer[id] = input->back_propagation[id];

	__syncthreads();

	if (id < size_output) {
		DTYPE resultat = input_buffer[0] * node->matrix[id * size_input];
		for (int i = 1; i < size_input; i++)
			resultat += input_buffer[i] * node->matrix[id * size_input + i];
	
		output->back_propagation[id] = resultat;
	}
}
__device__ void backPropagation(buffer * input, buffer * output, node * node, FC * fc, PCPTR * pcptr, int node_size) {
	switch (node->type) {
	case PERCEPTRON:
		backpropagation_perceptron(input, output, pcptr + node->indice);
		break;
	case FCONNECTION:
		backpropagation_fullconnection(input, output, fc + node->indice);
		break;
	}
}

__device__ void gradient(buffer * resultat, DTYPE * expected) {
	int id = threadIdx.x;
	if (id < resultat->size) {
		resultat->back_propagation[id] = resultat->propagation_valid[id] - expected[id];
	}
}

__global__ void gradient_kernel(buffer * resultat, DTYPE * expected) {
	int id = threadIdx.x;
	if (id < resultat->size) {
		resultat->back_propagation[id] = resultat->propagation_valid[id] - expected[id];
	}
}

__device__ void learn_perceptron(buffer * input, buffer * output, PCPTR * pcptr, DTYPE alpha) {
	int id = threadIdx.x;
	if (id < input->size)
		pcptr->bias[id] -= alpha * input->back_propagation[id];
}
__device__ void learn_fullconnection(buffer * input, buffer * output, FC * fc, DTYPE alpha) {
	int input_size = input->size;
	int output_size = output->size;

	int id = threadIdx.x ;
	
	__shared__ DTYPE input_prop[SHAREDSIZE / 2];
	__shared__ DTYPE output_backprop[SHAREDSIZE / 2];
	

	if (id < output_size)
		output_backprop[id] = output->back_propagation[id];

	if (id < input_size)
		input_prop[id % input_size] = input->propagation_valid[id % input_size];

	__syncthreads();
	if (id < output_size)
		for (int i = 0; i < input_size; i++)
			fc->matrix[id + output_size * i] -= alpha * input_prop[i] * output_backprop[id];
}
__device__ void learn(buffer * input, buffer * output, node * node, FC * fc, PCPTR * pcptr, DTYPE alpha) {
	switch (node->type) {
	case PERCEPTRON:
		learn_perceptron(input, output, pcptr + node->indice, alpha);
		break;
	case FCONNECTION:
		learn_fullconnection(input, output, fc + node->indice, alpha);
		break;
	}
}

__global__ void kernelNetwork_learningNoPipeline(buffer buffers[], node nodes[], FC fc[], PCPTR pcptr[], int node_size, float * expected, DTYPE alpha) {
	//propagation
	for (uint32_t i = 0; i < node_size; i++) {
		propagationNoPipeline(&(buffers[i]), &(buffers[i + 1]), &(nodes[i]), fc, pcptr);
		__syncthreads();
	} 
	//gradient
	gradient(buffers + node_size, expected);
	__syncthreads();

	//backpropagation
	for (int32_t i = node_size - 1; i >= 0; i--) {
		backPropagation(buffers + i + 1, buffers + i, nodes + i, fc, pcptr, node_size);
		__syncthreads();
	}

	//apprentissage
	for (uint32_t i = 0; i < node_size; i++) {
		learn(buffers + i, buffers + i + 1, nodes + i, fc, pcptr, alpha);
		__syncthreads();
	}
}

__global__ void kernelNetwork_propagationNoPipeline(buffer * buffers, node *nodes, FC *fc, PCPTR *pcptr, int node_size) {
	for (uint32_t i = 0; i < node_size; i++) {
		propagationNoPipeline(buffers + i, buffers + i + 1, nodes + i, fc, pcptr);
		__syncthreads();
	}
}

__global__ void kernelPerceptronLayer_propagation(buffer *input, buffer *output, PCPTR *node) {
	int id = threadIdx.x;
	output->propagation_valid[id] = ((DTYPE)1) / (((DTYPE)1) + ((DTYPE)__expf(-input->propagation_valid[id] - node->bias[id])));
}
__global__ void kernelFullConnection_propagation(buffer *input, buffer *output, FC *node) {
	int id = threadIdx.x;
	uint32_t input_size = input->size;
	uint32_t output_size = output->size;

	//__shared__ DTYPE input_buffer[SHAREDSIZE];

	//input_buffer[id] = input->propagation_valid[id];

	__syncthreads();

	/*A paraleliser*/
	DTYPE resultat = input->propagation_valid[0] * node->matrix[id];
	for (uint32_t i = 1; i < input_size; i++) {
		resultat += input->propagation_valid[i] * node->matrix[id + output_size * i];
	}

	output->propagation_valid[id] = resultat;
}

__global__ void kernelPerceptronLayer_backpropagation(buffer *input, buffer *output, PCPTR *node) {
	int id = threadIdx.x;
	output->back_propagation[id] = input->back_propagation[id] * input->propagation_valid[id] * ((DTYPE)1 - input->propagation_valid[id]);
}
__global__ void kernelFullConnection_backpropagation(buffer *input, buffer *output, FC *node) {
	int id = threadIdx.x;
	//__shared__ DTYPE input_buffer[SHAREDSIZE];
	//int size_output = output->size;
	int size_input = input->size;

	//int id_output = id % size_output;

	//if (id < size_input)
//	input_buffer[id % size_input] = input->back_propagation[id % size_input];

	__syncthreads();

	//if (id < size_output) {
	DTYPE resultat = input->back_propagation[0] * node->matrix[id * size_input];
	for (int i = 1; i < size_input; i++)
		resultat += input->back_propagation[i] * node->matrix[id * size_input + i];

	output->back_propagation[id] = resultat;

}

__global__ void kernelPerceptronLayer_learning(buffer * input, buffer * output, PCPTR * pcptr, DTYPE alpha) {
	int id = threadIdx.x;
	pcptr->bias[id] -= alpha * input->back_propagation[id];
}
__global__ void kernelFullConnection_learning(buffer * input, buffer * output, FC * fc, DTYPE alpha) {
	int input_size = input->size;
	int output_size = output->size;

	int id = threadIdx.x;
	/*
	__shared__ DTYPE input_prop[SHAREDSIZE / 2];
	__shared__ DTYPE output_backprop[SHAREDSIZE / 2];


	if (id < output_size)
		output_backprop[id] = output->back_propagation[id];

	if (id < input_size)
		input_prop[id % input_size] = input->propagation_valid[id % input_size];

	__syncthreads();
	if (id < output_size)*/
		for (int i = 0; i < input_size; i++)
			fc->matrix[id + output_size * i] -= alpha * input->propagation_valid[i] * output->back_propagation[id];
}


void propagationNoPipeline_kernelcall(int node_number, int maxdim, buffer * GPUbuffer_tab, node *GPUnode_tab, FC *GPUFC_tab, PCPTR *GPUPCPTR_tab) {
	kernelNetwork_propagationNoPipeline << <1, maxdim >> >(GPUbuffer_tab, GPUnode_tab, GPUFC_tab, GPUPCPTR_tab, node_number);
}

void learningNoPipeline_kernelcall(buffer * buffers, node * nodes, FC * fc, PCPTR * pcptr, int node_size, float * expected, DTYPE alpha, int maxdim) {
	kernelNetwork_learningNoPipeline <<<1, maxdim>>>(buffers, nodes, fc, pcptr, node_size, expected, alpha);
}

/*
	il faut que buffer soit decroissant
*/
void propagationNoPipeline_AtomicKernelcall(
	buffer * buffers, FC * fc, PCPTR * pcptr, 
	std::vector<type_node> &node_type, std::vector<uint32_t> &index, 
	std::vector<uint32_t> &buffer_size, int node_number) {
	
	for (int i = 0; i < node_number; i++) {
		switch (node_type[i]) {
		case FCONNECTION:
			//printf("kernel call : FC\n");
			kernelFullConnection_propagation <<<1, buffer_size[i + 1] >> >(buffers + i, buffers + i + 1, fc + index[i]);
			break;
		case PERCEPTRON:
			//printf("kernel call : PERCEP\n");
			kernelPerceptronLayer_propagation <<<1, buffer_size[i + 1] >> >(buffers + i, buffers + i + 1, pcptr + index[i]);
			break;
		}
	}
}

void learningNoPipeline_AtomicKernelcall(
	buffer * buffers, FC * fc, PCPTR * pcptr,
	std::vector<type_node> &node_type, std::vector<uint32_t> &index,
	std::vector<uint32_t> &buffer_size, int node_number, DTYPE * expected, DTYPE alpha) {
	
	propagationNoPipeline_AtomicKernelcall(buffers, fc, pcptr, node_type, index, buffer_size, node_number);

	gradient_kernel << <1, buffer_size[node_number] >> >(buffers + node_number, expected);

	for (int i = node_number - 1; i >= 0; i--) {
		switch (node_type[i]) {
		case FCONNECTION:
			//printf("kernel call : FC\n");
			kernelFullConnection_backpropagation << <1, buffer_size[i] >> >(buffers + i + 1, buffers + i, fc + index[i]);
			break;
		case PERCEPTRON:
			//printf("kernel call : PERCEP\n");
			kernelPerceptronLayer_backpropagation << <1, buffer_size[i] >> >(buffers + i + 1, buffers + i, pcptr + index[i]);
			break;
		}
	}

	for (int i = 0; i < node_number; i++) {
		switch (node_type[i]) {
		case FCONNECTION:
			//printf("kernel call : FC\n");
			kernelFullConnection_learning <<<1, buffer_size[i + 1] >> >(buffers + i + 1, buffers + i, fc + index[i], alpha);
			break;
		case PERCEPTRON:
			//printf("kernel call : PERCEP\n");
			kernelPerceptronLayer_learning <<<1, buffer_size[i + 1] >> >(buffers + i + 1, buffers + i, pcptr + index[i], alpha);
			break;
		}
	}
}
